
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void hello_from_gpu()
{
    printf("Hello World from the GPU!\n");
}



int main(void)
{
    // 核函数调用  1.线程块数量 2.线程块里面线程数量
    hello_from_gpu<<<3,1>>>();
    // 主机代码
    hipDeviceSynchronize();
    return 0;
}
